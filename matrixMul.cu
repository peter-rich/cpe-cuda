#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions provided by CUDA
#include "hip/hip_runtime_api.h"


#define TILE_WIDTH 16


__global__ void matrixMultiplier(float* d_A, float* d_B, float* d_C, int j, int k, int l) {

    // Allocate shared memory space
    __shared__ float A_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_shared[TILE_WIDTH][TILE_WIDTH];
    
    // Set block and thread position variables
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;

    // Detemine current C row and column
    int C_Row = block_y * TILE_WIDTH + thread_y;
    int C_Col = block_x * TILE_WIDTH + thread_x;

    float C_value = 0;
    // Determine number of phases required and start iteration
    int phase_count = ceil(k /(float)TILE_WIDTH);
    for (int phase = 0; phase < phase_count; ++phase) {
        if (C_Row < j && (phase * TILE_WIDTH + thread_x) < k) {
            // Load A value into shared mem
            A_shared[thread_y][thread_x] = d_A[C_Row * k + phase * TILE_WIDTH + thread_x];
        }
        else {
            A_shared[thread_y][thread_x] = 0.0;
        }

        if (C_Col < l && (phase * TILE_WIDTH + thread_y) < k) {
            // Load B value into shared mem
            B_shared[thread_y][thread_x] = d_B[(phase * TILE_WIDTH + thread_y) * l + C_Col];
        }
        else {
            B_shared[thread_y][thread_x] = 0.0;
        }
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++) {
            // Multiple A and B values and add to current C value
            C_value += A_shared[thread_y][i] * B_shared[i][thread_x];
        }
        __syncthreads();
    }
    if (C_Row < j && C_Col < l) {
        // Write C values to global memory
        d_C[C_Row * l + C_Col] = C_value;
    }
}

// Function to perform matrix multiplication on the CPU
void hostMatrixMultiplier(float* A, float* B, float* C, unsigned int row_Dim_A, unsigned int col_Dim_A, unsigned int col_Dim_B) {
    for (int c_row = 0; c_row < row_Dim_A; c_row++) {

        for (int c_col = 0; c_col < col_Dim_B; c_col++) {
            C[c_row * col_Dim_B + c_col] = 0;
            
            for (int i = 0; i < col_Dim_A; i++) {
                C[c_row * col_Dim_B + c_col] += A[c_row * col_Dim_A + i] * B[i * col_Dim_B + c_col];
            }
        }
    }
}

// Function to compare C matrix from GPU and C matrix from CPU
// Used to validate GPU's results
int checkResults(float* h_C, float* C, int size_C) {
    for (int i = 0; i < size_C; i++) {
        if (h_C[i] != C[i]) {
            printf("\nMatrices not equal!\n");
            return(1);
        }
    }
    printf("\nMatrices equal\n");
    return (0);
}



int main(int argc, char** argv) {

    size_t optind;
    int row_Dim_A = 2, col_Dim_A = 2, col_Dim_B = 2;
    // Check for input matrix sizes
    for (optind = 1; optind < argc && argv[optind][0] == '-'; optind++) {
        if (argv[optind][1] == 'i') {
            row_Dim_A = atoi(argv[optind + 1]);
            col_Dim_A = atoi(argv[optind + 2]);
            col_Dim_B = atoi(argv[optind + 3]);
        }
    }

    // Create Matrix A and B
    unsigned int size_A = row_Dim_A * col_Dim_A;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)(malloc(mem_size_A));

    unsigned int size_B = col_Dim_A * col_Dim_B;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)(malloc(mem_size_B));

    int row;

    // Fill Matricies
    srand(time(NULL));
    int i = 1;
    for (row = 0; row < size_A; row++) {
        h_A[row] = rand();
        i++;
    }

    i = 1;
    for (row = 0; row < size_B; row++) {
        h_B[row] = rand();
        i++;
    }


    /*
    // Print Matricies

    printf("Matrix A:");
    for (int i = 0; i < size_A; i++) {
        if (i % col_Dim_A == 0) {
            printf("\n");
        }
        printf("%f\t", h_A[i]);
    }
    printf("\n\n");

    printf("Matrix B:");
    for (int i = 0; i < size_B; i++) {
        if (i % col_Dim_B == 0) {
            printf("\n");
        }
        printf("%f\t", h_B[i]);
    }
    printf("\n\n");
    */


    // Declare device variables
    float* d_A, * d_B, * d_C;
    unsigned int size_C = row_Dim_A * col_Dim_B;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* h_C = (float*)(malloc(mem_size_C));;

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));


    // Allocate memory on the device
    checkCudaErrors(hipMalloc((void**)&d_A, mem_size_A));
    checkCudaErrors(hipMalloc((void**)&d_B, mem_size_B));
    checkCudaErrors(hipMalloc((void**)&d_C, mem_size_C));

    // Copy input matricies to device memory
    checkCudaErrors(hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice));

    dim3 DimGrid(ceil(col_Dim_B / (float)TILE_WIDTH) + 1, ceil(row_Dim_A / (float)TILE_WIDTH) + 1, 1), DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    printf("\nGrid Dimensions %d, %d: \n", DimGrid.x, DimGrid.y);
    printf("Block Dimensions %d, %d: \n\n", DimBlock.x, DimBlock.y);

    // Record the start event
    hipStream_t stream;
    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    checkCudaErrors(hipEventRecord(start, stream));
    
    int nIter = 150;
    for (int j = 0; j < nIter; j++) {
        matrixMultiplier << <DimGrid, DimBlock >> > (d_A, d_B, d_C, row_Dim_A, col_Dim_A, col_Dim_B);
    }

    // Record the stop event
    checkCudaErrors(hipEventRecord(stop, stream));

    // Wait for the stop event to complete
    checkCudaErrors(hipEventSynchronize(stop));

    float msecTotal = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    float flopsPerMatrixMul = 2.0 * static_cast<float>(col_Dim_A) *
        static_cast<float>(row_Dim_A) *
        static_cast<float>(col_Dim_B);
    float gigaFlops = (flopsPerMatrixMul * 1.0e-9f) /
        (msecPerMatrixMul / 1000.0f);
    printf("\nGPU Done\n");
    printf(
        "Performance= %.2f GFlop/s\n Time= %.3f msec\n Size= %.0f Ops\n" \
        " WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        DimBlock.x * DimBlock.y);



    // Copy result from device memory
    checkCudaErrors(hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost));
    // Free device memory
    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_C));

    /*
    printf("Matrix C:");
    for (int i = 0; i < size_C; i++) {
        if (i % col_Dim_B == 0) {
            printf("\n");
        }
        printf("%f\t", h_C[i]);
    }
    */

    float* C = (float*)(malloc(mem_size_C));;
    hostMatrixMultiplier(h_A, h_B, C, row_Dim_A, col_Dim_A, col_Dim_B);
    printf("\nCPU Done\n");


    /*
    printf("\nMatrix C:");
    for (int i = 0; i < size_C; i++) {
        if (i % col_Dim_B == 0) {
            printf("\n");
        }
        printf("%f\t", C[i]);
    }
    */


    checkResults(h_C, C, size_C);

    free(h_A);
    free(h_B);
    free(h_C);
}
